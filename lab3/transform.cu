#include <stdio.h>

#include "lodepng.h"
#include "transform.h"
#include "common.h"

int main(int argc, char* argv[]) {
    // Check for the command line argument
    if (argc != 3) {
        printf("Expected 2 arguments\n");
        return -1;
    }
    // The first argument is the program name, skip it
    // The next is the input PNG file
    char* inputName = argv[1];
    // The next is the output file name
    char* outputName = argv[2];
    // Now load the input PNG file
    unsigned char* image;
    unsigned width, height;
    unsigned readError = lodepng_decode32_file(&image, &width, &height, inputName);
    if (readError) {
        printf("Error when loading the input image: %s\n", lodepng_error_text(readError));
        return -1;
    }
    // Select the GPU first
    if (!selectBestGPU()) {
        printf("No CUDA supporting GPU found\n");
        return -1;
    }
    // Get the size of the output from the input
    unsigned outputWidth = width;
    unsigned outputHeight = height;
    getOutputSize(&outputWidth, &outputHeight);
    // Get the recommended block and grid sizes
    dim3 dimBlock;
    dim3 dimGrid;
    if (!findBestGridAndBlockDims2D(outputWidth, outputHeight, transform, &dimBlock, &dimGrid)) {
        printf("Could not calculate a suitable block size\n");
        return -1;
    }
    // Our image is made up of four 8 bit unsigned components
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(8, 8, 8, 8, hipChannelFormatKindUnsigned);
    // Allocate a CUDA array on the GPU to hold the input image
    hipArray* cuArray;
    hipMallocArray(&cuArray, &channelDesc, width, height);
    // Copy the image data to the GPU
    unsigned imageByteSize = width * height * sizeof(unsigned char) * 4;
    hipMemcpyToArray(cuArray, 0, 0, image, imageByteSize, hipMemcpyHostToDevice);
    // Delete the input image since we no longer need it
    free(image);
    // Create a resource description for the texture using the array
    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = cuArray;
    // Specify the texture object parameters
    struct hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] = hipAddressModeClamp;
    texDesc.addressMode[1] = hipAddressModeClamp;
    texDesc.filterMode = hipFilterModePoint;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = 0;
    // Create texture object
    hipTextureObject_t texture = 0;
    hipCreateTextureObject(&texture, &resDesc, &texDesc, NULL);
    // Allocate the output of transformation on the GPU
    unsigned imageOutputByteSize = outputWidth * outputHeight * sizeof(unsigned char) * 4;
    unsigned char* output;
    hipMalloc(&output, imageOutputByteSize);
    // Check for a CUDA error when creating the texture
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error %s\n", hipGetErrorString(error));
        return -1;
    }
    // Start the timer
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    // Invoke kernel
    transform<<<dimGrid, dimBlock>>>(output, texture, outputWidth, outputHeight);
    // Stop the time
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    // Get the time delta in miliseconds
    float elapsedMili;
    hipEventElapsedTime(&elapsedMili, start, stop);
    // Print the time taken
    printf("Took about %.4fms\n", elapsedMili);
    // Allocate some some CPU side memory for the output image
    unsigned char* imageOut = (unsigned char*) malloc(imageOutputByteSize);
    // Copy the output from the GPU back into the image
    hipMemcpy(imageOut, output, imageOutputByteSize, hipMemcpyDeviceToHost);
    // Destroy texture object
    hipDestroyTextureObject(texture);
    // Free device memory
    hipFreeArray(cuArray);
    hipFree(output);
    // Check for a CUDA error when finishing the job
    error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error %s\n", hipGetErrorString(error));
        return -1;
    }
    // Save the results
    unsigned outputError = lodepng_encode32_file(outputName, imageOut, outputWidth, outputHeight);
    if (outputError) {
        printf("Error when saving the output image: %s\n", lodepng_error_text(outputError));
        return -1;
    }
    return 0;
}
